#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

static __constant__ GpuData cData;

void SetKernelsGpuData()
{
    hipError_t status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

void GetKernelsGpuData()
{
    hipError_t status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy From cData failed");
}

uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

__global__ void kScaleAndBias_kernel(Float* pData, uint64_t size, Float scale, Float bias)
{
    uint64_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        Float value = pData[offset];
        pData[offset] = scale * value - bias;
    }
}

void kScaleAndBias(Float* pData, uint64_t size, Float scale, Float bias)
{
    const uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    const uint32_t blocks = CalculateBlocks(size);
    kScaleAndBias_kernel<<<blocks, threadsPerBlock>>>(pData, size, scale, bias);
    LAUNCHERROR("kScaleAndBias_kernel");
}

__global__ void kClearUnit_kernel(Float* pUnit, Float* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias[bpos];
    }
}

void kClearUnit(Float* pUnit, Float* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    LAUNCHERROR("kClearUnit_kernel");
}

__global__ void kClearDualSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos];
    }
}

void kClearDualSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearDualSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kClearDualSourceUnit_kernel");
}
__global__ void kClearTripleSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

void kClearTripleSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearTripleSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSourceUnit_kernel");
}
__global__ void kClearQuadSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

void kClearQuadSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearQuadSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSourceUnit_kernel");
}
__global__ void kLoadSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = (pDataWeight != NULL) ? pDataWeight[pos1] : (Float)1.0;
        uint64_t offset = pos * stride;
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            pUnit[pos2] = w;
        }
    }
}

void kLoadSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadSparseInputUnit failed");
    kLoadSparseInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadSparseInputUnit_kernel");
}
__global__ void kLoadIndexedSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1];
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = (pDataWeight != NULL) ? pDataWeight[pos1] : (Float)1.0;
        uint64_t offset = pos * stride;
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            pUnit[pos2] = w;
        }
    }
}

void kLoadIndexedSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadIndexedSparseInputUnit failed");
    kLoadIndexedSparseInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadIndexedSparseInputUnit_kernel");
}
template<typename T>
__global__ void kLoadSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = (pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0);
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * static_cast<Float>(data);
        }
    }
}

template<typename T>
void kLoadSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadSparseAnalogInputUnit failed");
    kLoadSparseAnalogInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadSparseAnalogInputUnit_kernel");
}
template<typename T>
__global__ void kLoadIndexedSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1];
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = (pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0);
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * static_cast<Float>(data);
        }
    }
}

template<typename T>
void kLoadIndexedSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadIndexedSparseAnalogInputUnit failed");
    kLoadIndexedSparseAnalogInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadIndexedSparseAnalogInputUnit_kernel");
}
__global__ void kLoadSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pRandom)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = cData._denoising_q * ((pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0));
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            Float value = pRandom[i];
            uint64_t pos2 = offset + pSparseIndex[i];
            if (value >= cData._denoising_p)
                pUnit[pos2] = w;
        }
    }
}

void kLoadSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadSparseDenoisedInputUnit failed");
    kLoadSparseDenoisedInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    LAUNCHERROR("kLoadSparseDenoisedInputUnit_kernel");
}
__global__ void kLoadIndexedSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pRandom)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = cData._denoising_q * ((pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0));
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            Float value = pRandom[i];
            uint64_t pos2 = offset + pSparseIndex[i];
            if (value >= cData._denoising_p)
                pUnit[pos2] = w;
        }
    }
}

void kLoadIndexedSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadIndexedSparseDenoisedInputUnit failed");
    kLoadIndexedSparseDenoisedInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    LAUNCHERROR("kLoadIndexedSparseDenoisedInputUnit_kernel");
}
template<typename T>
__global__ void kLoadSparseAnalogDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pRandom)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];
        Float w = cData._denoising_q * ((pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0));
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            Float value = pRandom[i];
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            if (value >= cData._denoising_p)
                pUnit[pos2] = w * data;
        }
    }
}

template<typename T>
void kLoadSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadSparseAnalogDenoisedInputUnit failed");
    kLoadSparseAnalogDenoisedInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    LAUNCHERROR("kLoadSparseAnalogDenoisedInputUnit_kernel");
}
template<typename T>
__global__ void kLoadIndexedSparseAnalogDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pRandom)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pIndex[pos + position];
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1] + threadIdx.x;
        uint64_t end = pSparseEnd[pos1];
        Float w = cData._denoising_q * ((pDataWeight != nullptr) ? pDataWeight[pos1] : static_cast<Float>(1.0));
        uint64_t offset = static_cast<uint64_t>(pos) * static_cast<uint64_t>(stride);
        for (uint64_t i = start; i < end; i += blockDim.x)
        {
            Float value = pRandom[i];
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            if (value >= cData._denoising_p)
                pUnit[pos2] = w * data;
        }
    }
}

template<typename T>
void kLoadIndexedSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(Float));
    RTERROR(status, "kLoadIndexedSparseAnalogDenoisedInputUnit failed");
    kLoadIndexedSparseAnalogDenoisedInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    LAUNCHERROR("kLoadIndexedSparseAnalogDenoisedInputUnit_kernel");
}
template<typename T>
__global__ void kLoadInputUnit_kernel(uint32_t position, uint32_t stride, Float* pUnit, T* pData)
{
    uint32_t blockIdx_x = blockIdx.x;
    uint64_t pos = (blockIdx_x * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1 = blockIdx_x + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t soffset = static_cast<uint64_t>(pos1) * static_cast<uint64_t>(stride) + pos;
        uint64_t doffset = static_cast<uint64_t>(blockIdx_x) * static_cast<uint64_t>(stride) + pos;
        pUnit[doffset] = static_cast<Float>(pData[soffset]);
    }
}
__global__ void kLoadNormalizedInputUnit_kernel(int position, int stride, Float* pUnit, unsigned char* pData)
{
    int tid = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (tid < stride)
    {
        int pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : (blockIdx.x + position);
        int offset = blockIdx.x * stride + tid;
        pUnit[offset] = static_cast<Float>(pData[pos1 * stride + tid]) * cData._inv256 - cData._half;
    }
}
__global__ void kLoadNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, Float* pUnit, unsigned char* pData)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : (blockIdx.x + position);
        uint64_t soffset = pos1 * stride + pos;
        uint64_t doffset = blockIdx.x * stride + pos;
        pUnit[doffset] = static_cast<Float>(pData[soffset]) * cData._inv128;
    }
}

template<typename T>
void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, T* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    LAUNCHERROR("kLoadInputUnit_kernel");
}

template<>
void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, unsigned char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    LAUNCHERROR("kLoadNormalizedInputUnit_kernel");
}
template<typename T>
__global__ void kLoadIndexedInputUnit_kernel(uint32_t position, uint32_t stride, Float* pUnit, uint32_t* pIndex, T* pData, bool bShuffleIndices, uint32_t* pShuffleIndex)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < stride)
    {
        uint32_t pos1 = pIndex[bShuffleIndices ? pShuffleIndex[blockIdx.y + position] : blockIdx.y + position];
        uint64_t soffset = pos1 * stride + tid;
        uint64_t doffset = blockIdx.y * stride + tid;
        pUnit[doffset] = pData[soffset];
    }
}
__global__ void kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, Float* pUnit, uint32_t* pIndex, unsigned char* pData, bool bShuffleIndices, uint32_t* pShuffleIndex)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < stride)
    {
        uint32_t pos1 = pIndex[bShuffleIndices ? pShuffleIndex[blockIdx.y + position] : blockIdx.y + position];
        uint64_t soffset = pos1 * stride + tid;
        uint64_t doffset = blockIdx.y * stride + tid;
        unsigned char pixel = pData[soffset];
        pUnit[doffset] = static_cast<Float>(pixel) * static_cast<Float>(1.0 / 256.0) - static_cast<Float>(0.5);
    }
}
template<typename T>
__global__ void kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, Float* pUnit, uint32_t* pIndex, T* pData)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < stride)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.y + position] : blockIdx.y + position];
        uint64_t soffset = pos1 * stride + tid;
        uint64_t doffset = blockIdx.y * stride + tid;
        pUnit[doffset] = static_cast<Float>(pData[soffset]) * static_cast<Float>(1.0 / 128.0);
    }
}

template<typename T>
void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, Float* pUnit, uint32_t* pIndex, T* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadIndexedNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);
    LAUNCHERROR("kLoadIndexedNormalizedInputUnit_kernel");
}
__global__ void kAddBias_kernel(Float* pUnit, Float* pBias, uint32_t stride, uint32_t size)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        uint32_t bpos = tid % stride;
        pUnit[tid] += pBias[bpos];
    }
}

void kAddBias(Float* pUnit, Float* pBias, uint32_t stride, uint32_t batch)
{
    uint32_t size = stride * batch;
    uint32_t blocks = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias, stride, size);
    LAUNCHERROR("kAddBias_kernel");
}
__global__ void kAddDualBias_kernel(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        uint32_t bpos = tid % stride;
        pUnit[tid] += pBias1[bpos] + pBias2[bpos];
    }
}

void kAddDualBias(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t blocks = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddDualBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kAddDualBias_kernel");
}
__global__ void kAddTripleBias_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        uint32_t bpos = tid % stride;
        pUnit[tid] += pBias1[bpos] + pBias2[bpos] + pBias3[tid];
    }
}

void kAddTripleBias(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t blocks = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddTripleBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kAddTripleBias_kernel");
}
__global__ void kAddQuadBias_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] += pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

void kAddQuadBias(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kAddQuadBias_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kAddQuadBias_kernel");
}
#if (__CUDA_ARCH__ >= 600)
static const uint32_t MAXSPARSE = SM_6X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_6X_MAXSPARSEANALOG;
#elif (__CUDA_ARCH__ >= 500)
static const uint32_t MAXSPARSE = SM_5X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_5X_MAXSPARSEANALOG;
#else
static const uint32_t MAXSPARSE = SM_3X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_3X_MAXSPARSEANALOG;
#endif
__global__ void kCalculateSparseZ_kernel(uint32_t position, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pUnit, Float beta)
{
    __shared__ uint32_t sOffset[MAXSPARSE];

    position = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
    uint64_t start = pSparseStart[position];
    uint64_t end = pSparseEnd[position];
    Float w = (pDataWeight != NULL) ? pDataWeight[position] : (Float)1.0;
    pUnit += blockIdx.x * stride;

    while (start < end)
    {
        uint32_t inputs = min(static_cast<uint32_t>(end - start), static_cast<uint32_t>(MAXSPARSE));
        uint64_t tend = start + inputs;
        uint64_t tstart = start + threadIdx.x;
        uint32_t pos = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos] = pSparseIndex[tstart] * stride;
            pos += blockDim.x;
            tstart += blockDim.x;
        }

        __syncthreads();

        uint32_t tgx = threadIdx.x & cData._warpMask;
        uint32_t opos = threadIdx.x - tgx;

        while (opos < stride)
        {
            opos += tgx;

            if (opos < stride)
            {
                Float unit = (beta == static_cast<Float>(0.0)) ? static_cast<Float>(0.0) : (beta * pUnit[opos]);

                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit += w * pWeight[offset + opos];
                }

                pUnit[opos] = unit;
            }

            opos -= tgx;

            if (tgx == 0)
            {
                opos = atomicAdd(&sOffset[0], cData._warpSize);
            }

            opos = __shfl_sync(0xFFFFFFFF, opos, 0);
        }

        start = tend;

        __syncthreads();

        if (start < end)
        {
            beta = static_cast<Float>(1.0);
        }
    }
}

void kCalculateSparseZ(uint32_t position, uint32_t batch, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pUnit, Float beta)
{
    uint32_t threads = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pUnit, beta);
    LAUNCHERROR("kCalculateSparseZ_kernel");
}
__global__ void kCalculateIndexedSparseZ_kernel(uint32_t position, uint32_t stride, Float* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pUnit, Float beta)
{
    __shared__ uint32_t sOffset[MAXSPARSE];

    uint32_t sOpos = blockDim.x;
    position = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
    uint64_t start = pSparseStart[position];
    uint64_t end = pSparseEnd[position];
    Float w = (pDataWeight != NULL) ? pDataWeight[position] : static_cast<Float>(1.0);
    pUnit += blockIdx.x * stride;

    while (start < end)
    {
        uint32_t inputs = static_cast<uint32_t>(min(static_cast<uint64_t>(end - start), static_cast<uint64_t>(MAXSPARSE)));
        uint64_t tend = start + inputs;
        uint64_t tstart = start + threadIdx.x;
        uint32_t pos = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos] = pSparseIndex[tstart] * stride;
            pos += blockDim.x;
            tstart += blockDim.x;
        }

        __syncthreads();

        uint32_t tgx = threadIdx.x & cData._warpMask;
        uint32_t opos = threadIdx.x - tgx;

        while (opos < stride)
        {
            opos += tgx;

            if (opos < stride)
            {
                Float unit = (beta == static_cast<Float>(0.0)) ? static_cast<Float>(0.0) : (beta * pUnit[opos]);

                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit += w * pWeight[offset + opos];
                }

                pUnit[opos] = unit;
            }

            opos -= tgx;

            if (tgx == 0)
            {
                opos = atomicAdd(&sOpos, cData._warpSize);
            }

            opos = __shfl_sync(0xFFFFFFFF, opos, 0);
        }

        start = tend;

        __syncthreads();

        if (start < end)
        {
            beta = static_cast<Float>(1.0);
        }
    }
}

void kCalculateIndexedSparseZ(uint32_t position, uint32_t batch, uint32_t stride, Float* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, Float* pUnit, Float beta)
{
    uint32_t threads = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseZ_kernel");
}
template<typename T>
__global__ void LAUNCH_BOUNDS256() kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pUnit, Float beta)
{
    __shared__ uint32_t sOpos;
    __shared__ uint32_t sIndex[MAXSPARSEANALOG];
    __shared__ T sData[MAXSPARSEANALOG];

    sOpos = blockDim.x;
    position = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;

    uint64_t start = pSparseStart[position];
    uint64_t end = pSparseEnd[position];
    Float w = (pDataWeight != NULL) ? pDataWeight[position] : (Float)1.0;
    pUnit += blockIdx.x * stride;

    while (start < end)
    {
        sOpos = blockDim.x;
        uint32_t inputs = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend = start + inputs;
        uint64_t tstart = start + threadIdx.x;
        uint32_t pos = threadIdx.x;

        // Load sparse index and data into shared memory
        while (tstart < tend)
        {
            sIndex[pos] = pSparseIndex[tstart];
            sData[pos] = w * pSparseData[tstart];
            pos += blockDim.x;
            tstart += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        uint32_t tgx = threadIdx.x & cData._warpMask;
        uint32_t opos = threadIdx.x - tgx;
        while (opos < stride)
        {
            opos += tgx;
            if (opos < stride)
            {
                Float unit = (beta == (Float)0.0) ? (Float)0.0 : (beta * pUnit[opos]);

                // Compute weighted sum using shared memory
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sIndex[i] * stride;
                    unit += pWeight[offset + opos] * sData[i];
                }

                pUnit[opos] = unit;
            }
            opos -= tgx;

            if (tgx == 0)
            {
                opos = atomicAdd(&sOpos, cData._warpSize);
            }
            opos = __shfl_sync(0xffffffff, opos, 0);
        }

        start = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta = (Float)1.0;
    }
}
template<>
__global__ void LAUNCH_BOUNDS256() kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, unsigned char* pSparseData, Float* pUnit, Float beta)
{
    __shared__ uint32_t sOpos;
    __shared__ uint32_t sIndex[MAXSPARSEANALOG];
    __shared__ Float sValue[MAXSPARSEANALOG];

    sOpos = blockDim.x;
    position = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;

    uint64_t start = pSparseStart[position];
    uint64_t end = pSparseEnd[position];
    Float w = (pDataWeight != NULL) ? pDataWeight[position] : (Float)1.0;
    pUnit += blockIdx.x * stride;

    while (start < end)
    {
        sOpos = blockDim.x;
        uint32_t inputs = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend = start + inputs;
        uint64_t tstart = start + threadIdx.x;
        uint32_t pos = threadIdx.x;

        // Load sparse index and data into shared memory
        while (tstart < tend)
        {
            sIndex[pos] = pSparseIndex[tstart];
            sValue[pos] = w * ((Float)pSparseData[tstart] * (Float)(1.0 / 256.0));
            pos += blockDim.x;
            tstart += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        uint32_t tgx = threadIdx.x & cData._warpMask;
        uint32_t opos = threadIdx.x - tgx;
        while (opos < stride)
        {
            opos += tgx;
            if (opos < stride)
            {
                Float unit = (beta == (Float)0.0) ? (Float)0.0 : (beta * pUnit[opos]);

                // Compute weighted sum using shared memory
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sIndex[i] * stride;
                    unit += pWeight[offset + opos] * sValue[i];
                }

                pUnit[opos] = unit;
            }
            opos -= tgx;

            if (tgx == 0)
            {
                opos = atomicAdd(&sOpos, cData._warpSize);
            }
            opos = __shfl_sync(0xffffffff, opos, 0);
        }

        start = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta = (Float)1.0;
    }
}
template<>
__global__ void LAUNCH_BOUNDS256() kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, char* pSparseData, Float* pUnit, Float beta)
{
    __shared__ uint32_t sOpos;
    __shared__ uint32_t sOffset[MAXSPARSEANALOG];
    __shared__ Float sValue[MAXSPARSEANALOG];

    sOpos = blockDim.x;
    position = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
    uint64_t start = pSparseStart[position];
    uint64_t end = pSparseEnd[position];
    Float w = (pDataWeight != NULL) ? pDataWeight[position] : 1.0f;
    pUnit += blockIdx.x * stride;

    while (start < end)
    {
        sOpos = blockDim.x;
        uint32_t inputs = ullmin(end - start, static_cast<uint64_t>(MAXSPARSEANALOG));
        uint64_t tend = start + inputs;
        uint64_t tstart = start + threadIdx.x;
        uint32_t pos = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos] = pSparseIndex[tstart] * stride;
            sValue[pos] = w * static_cast<Float>(pSparseData[tstart]) * (1.0f / 256.0f);
            pos += blockDim.x;
            tstart += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        uint32_t tgx = threadIdx.x & cData._warpMask;
        uint32_t opos = threadIdx.x - tgx;

        while (opos < stride)
        {
            opos += tgx;
            if (opos < stride)
            {
                Float unit = (beta == 0.0f) ? 0.0f : beta * pUnit[opos];
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit = fmaf(pWeight[opos + offset], sValue[i], unit);
                }

                pUnit[opos] = unit;
            }
            opos -= tgx;

            if (tgx == 0)
            {
                opos = atomicAdd(&sOpos, cData._warpSize);
            }
            opos = SHFL(opos, 0);
        }

        start = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
    }
}

template<typename T>
void kCalculateSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, Float* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, Float* pDataWeight, T* pSparseData, Float* pUnit, Float beta)
{
    uint32_t threads = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogZ_kernel");
}
