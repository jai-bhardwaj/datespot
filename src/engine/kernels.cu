#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

static __constant__ GpuData cData;

void SetKernelsGpuData()
{
    hipError_t status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

void GetKernelsGpuData()
{
    hipError_t status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy From cData failed");
}

uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

__global__ void kScaleAndBias_kernel(Float* pData, uint64_t size, Float scale, Float bias)
{
    uint64_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        Float value = pData[offset];
        pData[offset] = scale * value - bias;
    }
}

void kScaleAndBias(Float* pData, uint64_t size, Float scale, Float bias)
{
    const uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    const uint32_t blocks = CalculateBlocks(size);
    kScaleAndBias_kernel<<<blocks, threadsPerBlock>>>(pData, size, scale, bias);
    LAUNCHERROR("kScaleAndBias_kernel");
}

__global__ void kClearUnit_kernel(Float* pUnit, Float* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias[bpos];
    }
}

void kClearUnit(Float* pUnit, Float* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    LAUNCHERROR("kClearUnit_kernel");
}

__global__ void kClearDualSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos];
    }
}

void kClearDualSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearDualSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kClearDualSourceUnit_kernel");
}
__global__ void kClearTripleSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

void kClearTripleSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearTripleSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSourceUnit_kernel");
}
__global__ void kClearQuadSourceUnit_kernel(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

void kClearQuadSourceUnit(Float* pUnit, Float* pBias1, Float* pBias2, Float* pBias3, Float* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = CalculateBlocks(size);
    kClearQuadSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSourceUnit_kernel");
}
