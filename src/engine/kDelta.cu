#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "NNTypes.h"
#include <limits>
#include <hip/hip_fp16.h>

static __constant__ GpuData cData;

void SetKDeltaGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKDeltaGpuData copy to cData failed");
}

void GetKDeltaGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));     
    RTERROR(status, "hipMemcpyFromSymbol: GetKDeltaGpuData copy From cData failed");
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateSigmoidOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, NNFloat* pDelta, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * static_cast<NNFloat>(1.0 / (T(1) << 8 * sizeof(T)));
        pDelta[uOffset + pos] = w * (a - t) * a * (static_cast<NNFloat>(1.0) - a);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateSigmoidOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, NNFloat* pDelta, uint8_t* pData, NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(UINT8_MAX));
        pDelta[uOffset + pos] = w * (a - t) * a * (static_cast<NNFloat>(1.0) - a);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateSigmoidOutputDelta_kernel<int8_t>(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, NNFloat* pDelta, int8_t* pData, NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(INT8_MAX));
        pDelta[uOffset + pos] = w * (a - t) * a * (static_cast<NNFloat>(1.0) - a);
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateTanhOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, NNFloat* pDelta, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t) * (static_cast<NNFloat>(1.0) - a * a);
    }
}


#include <hip/hip_fp16.h>

template <>
__global__ void LAUNCH_BOUNDS() kCalculateTanhOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(UINT8_MAX));
        pDelta[uOffset + pos] = w * (a - t) * (static_cast<NNFloat>(1.0) - a * a);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateTanhOutputDelta_kernel<int8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const int8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(INT8_MAX));
        pDelta[uOffset + pos] = w * (a - t) * (static_cast<NNFloat>(1.0) - a * a);
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateLinearOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateLinearOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(UINT8_MAX));
        pDelta[uOffset + pos] = w * (a - t);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateLinearOutputDelta_kernel<int8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const int8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(INT8_MAX));
        pDelta[uOffset + pos] = w * (a - t);
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateRELUOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t) * (a > static_cast<NNFloat>(0.0));
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateRELUOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(UINT8_MAX));
        pDelta[uOffset + pos] = w * (a - t) * (a > static_cast<NNFloat>(0.0));
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateRELUOutputDelta_kernel<char>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const char* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(128.0));
        pDelta[uOffset + pos] = w * (a - t) * (a > static_cast<NNFloat>(0.0));
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateLRELUOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight, NNFloat slope)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t) * (a > static_cast<NNFloat>(0.0)) + (a <= static_cast<NNFloat>(0.0)) * slope;
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateLRELUOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight, NNFloat slope)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(256.0));
        pDelta[uOffset + pos] = w * (a - t) * (a > static_cast<NNFloat>(0.0)) + (a <= static_cast<NNFloat>(0.0)) * slope;
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateELUOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight, NNFloat alpha)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t) * (a >= static_cast<NNFloat>(0.0)) + (a < static_cast<NNFloat>(0.0)) * (a + alpha);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateELUOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight, NNFloat alpha)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(256.0));
        pDelta[uOffset + pos] = w * (a - t) * (a >= static_cast<NNFloat>(0.0)) + (a < static_cast<NNFloat>(0.0)) * (a + alpha);
    }
}

#include <hip/hip_fp16.h>

template <>
__global__ void LAUNCH_BOUNDS() kCalculateELUOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const char* pData, const NNFloat* pDataWeight, NNFloat alpha)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(128.0));
        pDelta[uOffset + pos] = w * (a - t) * ((a >= static_cast<NNFloat>(0.0)) + (a < static_cast<NNFloat>(0.0)) * (a + alpha));
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateSELUOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight, NNFloat alpha, NNFloat lambda)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t) * ((a >= static_cast<NNFloat>(0.0)) * lambda + (a < static_cast<NNFloat>(0.0)) * (lambda * alpha * expf(a)));
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateSELUOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight, NNFloat alpha, NNFloat lambda)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(256.0));
        pDelta[uOffset + pos] = w * (a - t) * ((a >= static_cast<NNFloat>(0.0)) * lambda + (a < static_cast<NNFloat>(0.0)) * (lambda * alpha * expf(a)));
    }
}

template <typename T>
__global__ void LAUNCH_BOUNDS() kCalculateSoftMaxOutputDelta_kernel(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const T* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = pData[dOffset + pos];
        pDelta[uOffset + pos] = w * (a - t);
    }
}

template <>
__global__ void LAUNCH_BOUNDS() kCalculateSoftMaxOutputDelta_kernel<uint8_t>(uint32_t position, uint32_t batch, uint32_t stride, const NNFloat* pUnit, NNFloat* pDelta, const uint8_t* pData, const NNFloat* pDataWeight)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint64_t uOffset = blockIdx.x * stride;
        uint64_t dpos = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset = dpos * stride;
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[dpos] : static_cast<NNFloat>(1.0);
        NNFloat a = pUnit[uOffset + pos];
        NNFloat t = static_cast<NNFloat>(pData[dOffset + pos]) * (static_cast<NNFloat>(1.0) / static_cast<NNFloat>(256.0));
        pDelta[uOffset + pos] = w * (a - t);
    }
}


