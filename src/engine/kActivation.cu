#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

static __constant__ GpuData cData;

/**
 * @brief Computes the atomic maximum of a floating-point value.
 *
 * @param address Pointer to the value to update.
 * @param val The value to compare and update.
 * @return The previous value before the update.
 */
__device__ inline float atomicMax(float* address, float val)
{
    int* address_as_i = reinterpret_cast<int*>(address);
    int old = *address_as_i;
    int assumed;
    do 
    {
        assumed = old;
        old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } 
    while (assumed != old);
    return __int_as_float(old);
}

/**
 * @brief Copies GpuData to the constant memory symbol cData.
 */
void SetKActivationGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKActivationGpuData copy to cData failed");
}

/**
 * @brief Copies GpuData from the constant memory symbol cData.
 */
void GetKActivationGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));     
    RTERROR(status, "hipMemcpyFromSymbol: GetKActivationGpuData copy From cData failed");
}

/**
 * @brief Kernel function to compute sigmoid activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateSigmoidActivation_kernel(Float* pData, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        Float a = 1.0f / (1.0f + expf(-pData[pos]));
        pData[pos] = a;
    }
}

/**
 * @brief Computes sigmoid activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
void kCalculateSigmoidActivation(Float* pData, uint64_t size)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateSigmoidActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateSigmoidActivation_kernel");
}

/**
 * @brief Kernel function to compute hyperbolic tangent activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateTanhActivation_kernel(Float* pData, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        pData[pos] = tanhf(pData[pos]);
}

/**
 * @brief Computes hyperbolic tangent activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
void kCalculateTanhActivation(Float* pData, uint64_t size)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateTanhActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateTanhActivation_kernel");
}

/**
 * @brief Kernel function to compute ReLU activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateRELUActivation_kernel(Float* pData, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        pData[pos] = fmaxf(0.0f, pData[pos]);
}

/**
 * @brief Computes ReLU activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 */
void kCalculateRELUActivation(Float* pData, uint64_t size)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateRELUActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateRELUActivation_kernel");
}

/**
 * @brief Kernel function to compute Leaky ReLU activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param slope Slope of the activation function for negative input values.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateLRELUActivation_kernel(Float* pData, uint64_t size, Float slope)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        Float val = pData[pos];
        pData[pos] = fmaxf(val, val * slope);
    }
}

/**
 * @brief Computes Leaky ReLU activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param slope Slope of the activation function for negative input values.
 */
void kCalculateLRELUActivation(Float* pData, uint64_t size, Float slope)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateLRELUActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size, slope);
    LAUNCHERROR("kCalculateLRELUActivation_kernel");
}

/**
 * @brief Kernel function to compute ELU activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param alpha Alpha value for the activation function.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateELUActivation_kernel(Float* pData, uint64_t size, Float alpha)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {   
        Float x = pData[pos];
        pData[pos] = (x > 0.0f) ? x : alpha * (expf(x) - 1.0f);
    }
}

/**
 * @brief Computes ELU activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param alpha Alpha value for the activation function.
 */
void kCalculateELUActivation(Float* pData, uint64_t size, Float alpha)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateELUActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size, alpha);
    LAUNCHERROR("kCalculateELUActivation_kernel");
}

/**
 * @brief Kernel function to compute SELU activation for each element in pData.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param alpha Alpha value for the activation function.
 * @param lambda Lambda value for the activation function.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateSELUActivation_kernel(Float* pData, uint64_t size, Float alpha, Float lambda)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {   
        Float x = pData[pos];
        pData[pos] = (x > 0.0f) ? lambda * x : lambda * alpha * (expf(x) - 1.0f);
    }
}

/**
 * @brief Computes SELU activation for the given data array on the GPU.
 *
 * @param pData Pointer to the data array.
 * @param size Number of elements in the array.
 * @param alpha Alpha value for the activation function.
 * @param lambda Lambda value for the activation function.
 */
void kCalculateSELUActivation(Float* pData, uint64_t size, Float alpha, Float lambda)
{
    uint32_t blocks = CalculateBlocks(size);
    kCalculateSELUActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size, alpha, lambda);
    LAUNCHERROR("kCalculateSELUActivation_kernel");
}

/**
 * @brief Kernel function to compute softmax activation for each element in pData with a given stride.
 *
 * @param pData Pointer to the data array.
 * @param stride The stride between elements in pData.
 */
__global__ void LAUNCH_BOUNDS()
kCalculateSoftMaxActivation_kernel(Float* pData, uint32_t stride)
{
    __shared__ unsigned long long int sAccumulator;
    __shared__ Float sMaxValue;

    if (threadIdx.x == 0)
    {
        sAccumulator = 0;
        sMaxValue = -std::numeric_limits<Float>::infinity();
    }
    __syncthreads();
    

    pData += blockIdx.x * stride;
    uint32_t pos = threadIdx.x;
    Float maxValue = -std::numeric_limits<Float>::infinity();
    
    while (pos < stride)
    {
        Float z = pData[pos];
        maxValue = fmaxf(z, maxValue);
        pos += blockDim.x;
    }
    
    uint32_t tgx = threadIdx.x & cData._warpMask;    
    maxValue = fmaxf(maxValue, __shfl_xor_sync(0xffffffff, maxValue, tgx ^ 1));
    maxValue = fmaxf(maxValue, __shfl_xor_sync(0xffffffff, maxValue, tgx ^ 2));
    maxValue = fmaxf(maxValue, __shfl_xor_sync(0xffffffff, maxValue, tgx ^ 4));
    maxValue = fmaxf(maxValue, __shfl_xor_sync(0xffffffff, maxValue, tgx ^ 8));
    maxValue = fmaxf(maxValue, __shfl_xor_sync(0xffffffff, maxValue, tgx ^ 16));

    if (tgx == 0) 
        atomicMax(&sMaxValue, maxValue);
    __syncthreads();        
    maxValue = sMaxValue;       

    pos = threadIdx.x;
    Float sum = 0.0f;
    while (pos < stride)
    {
        Float z = pData[pos];
        sum += expf(z - maxValue);
        pos += blockDim.x;
    }    
         
    sum += __shfl_xor_sync(0xffffffff, sum, tgx ^ 1);
    sum += __shfl_xor_sync(0xffffffff, sum, tgx ^ 2);
    sum += __shfl_xor_sync(0xffffffff, sum, tgx ^ 4);
    sum += __shfl_xor_sync(0xffffffff, sum, tgx ^ 8);
    sum += __shfl_xor_sync(0xffffffff, sum, tgx ^ 16);
    unsigned long long int lsum = llitoulli(llrintf(ERRORSCALEF * sum));
    if (tgx == 0) 
        atomicAdd(&sAccumulator, lsum);
    __syncthreads();               
    Float norm = 1.0f / static_cast<Float>((double)sAccumulator * ONEOVERERRORSCALE);
    

    pos = threadIdx.x;
    while (pos < stride)
    {
        Float z = pData[pos];
        Float a = expf(z - maxValue);
        pData[pos] = fminf(1.0f, a * norm);
        pos += blockDim.x;
    }    
}

/**
 * @brief Computes softmax activation for the given data array on the GPU with a specified batch size and stride.
 *
 * @param pData Pointer to the data array.
 * @param batch Number of batches in the data array.
 * @param stride The stride between elements in pData.
 */
void kCalculateSoftMaxActivation(Float* pData, uint32_t batch, uint32_t stride)
{
    uint32_t warps = getGpu()._threadsPerBlock / getGpu()._warpSize;
    kCalculateSoftMaxActivation_kernel<<<batch, getGpu()._threadsPerBlock>>>(pData, stride);
    LAUNCHERROR("kCalculateSoftMaxActivation_kernel");
}
