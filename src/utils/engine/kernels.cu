#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

/**
 * @brief Constant data stored on the GPU.
 */
static __constant__ GpuData cData;

/**
 * @brief Copies the GPU data to the constant symbol cData.
 */
void SetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

/**
 * @brief Copies the GPU data from the constant symbol cData.
 */
void GetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy from cData failed");
}

/**
 * @brief Calculates the number of blocks required for the given size.
 *
 * @param size The total number of elements.
 * @return The number of blocks needed to process the elements.
 */
uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

/**
 * @brief CUDA kernel to scale and bias an array in parallel.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
__global__ void kScaleAndBias_kernel(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint64_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        NNFloat value = pData[offset];
        pData[offset] = scale * value - bias;
    }
}

/**
 * @brief Applies scale and bias to an array using CUDA.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
void kScaleAndBias(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kScaleAndBias_kernel<<<blocks, threadsPerBlock>>>(pData, size, scale, bias);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a unit by assigning bias values in parallel.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearUnit_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] = pBias[bpos];
    }
}

/**
 * @brief Clears a unit using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearUnit(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        // Handle or report the CUDA kernel launch error
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a dual source unit by summing bias values in parallel.
 *
 * This kernel function sets the values of a dual source unit by summing the corresponding bias values from two different bias arrays.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearDualSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;

    if (pos < size)
    {
        pUnit[pos] = pBias1[bpos] + pBias2[bpos];
    }
}

/**
 * @brief Clears a dual source unit by summing bias values in parallel.
 *
 * This function clears a dual source unit by summing the bias values from two different bias arrays in parallel using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearDualSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);

    uint32_t threadsPerBlock = 256;

    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    kClearDualSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to compute the triple source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearTripleSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

/**
 * @brief Compute the triple source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearTripleSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearTripleSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSource_kernel");
}

/**
 * @brief CUDA kernel to compute the quad source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearQuadSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

/**
 * @brief Compute the quad source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearQuadSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearQuadSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSource_kernel");
}

/**
 * @brief CUDA kernel to load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
__global__ void kLoadSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];

        __shared__ NNFloat weight;
        if (threadIdx.x == 0)
        {
            weight = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        }
        __syncthreads();

        uint64_t offset = pos * stride;

        for (uint64_t i = threadIdx.x; i < (end - start); i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[start + i];
            pUnit[pos2] = weight;
        }
    }
}

/**
 * @brief Load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
void kLoadSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseInputUnit failed");
    kLoadSparseInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadSparseInputUnit_kernel");
}