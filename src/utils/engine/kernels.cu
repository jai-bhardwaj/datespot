#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

/**
 * @brief Constant data stored on the GPU.
 */
static __constant__ GpuData cData;

/**
 * @brief Copies the GPU data to the constant symbol cData.
 */
void SetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

/**
 * @brief Copies the GPU data from the constant symbol cData.
 */
void GetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy from cData failed");
}

/**
 * @brief Calculates the number of blocks required for the given size.
 *
 * @param size The total number of elements.
 * @return The number of blocks needed to process the elements.
 */
uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

/**
 * @brief CUDA kernel to scale and bias an array in parallel.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
__global__ void kScaleAndBias_kernel(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint64_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        NNFloat value = pData[offset];
        pData[offset] = scale * value - bias;
    }
}

/**
 * @brief Applies scale and bias to an array using CUDA.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
void kScaleAndBias(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kScaleAndBias_kernel<<<blocks, threadsPerBlock>>>(pData, size, scale, bias);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a unit by assigning bias values in parallel.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearUnit_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] = pBias[bpos];
    }
}

/**
 * @brief Clears a unit using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearUnit(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        // Handle or report the CUDA kernel launch error
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a dual source unit by summing bias values in parallel.
 *
 * This kernel function sets the values of a dual source unit by summing the corresponding bias values from two different bias arrays.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearDualSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;

    if (pos < size)
    {
        pUnit[pos] = pBias1[bpos] + pBias2[bpos];
    }
}

/**
 * @brief Clears a dual source unit by summing bias values in parallel.
 *
 * This function clears a dual source unit by summing the bias values from two different bias arrays in parallel using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearDualSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);

    uint32_t threadsPerBlock = 256;

    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    kClearDualSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to compute the triple source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearTripleSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

/**
 * @brief Compute the triple source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearTripleSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearTripleSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSource_kernel");
}

/**
 * @brief CUDA kernel to compute the quad source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearQuadSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

/**
 * @brief Compute the quad source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearQuadSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearQuadSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSource_kernel");
}

/**
 * @brief CUDA kernel to load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
__global__ void kLoadSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];

        __shared__ NNFloat weight;
        if (threadIdx.x == 0)
        {
            weight = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        }
        __syncthreads();

        uint64_t offset = pos * stride;

        for (uint64_t i = threadIdx.x; i < (end - start); i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[start + i];
            pUnit[pos2] = weight;
        }
    }
}

/**
 * @brief Load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
void kLoadSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseInputUnit failed");
    kLoadSparseInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadSparseInputUnit_kernel");
}

/**
 * @brief CUDA kernel for loading indexed sparse input units.
 *
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride of the input.
 * @param pUnit Pointer to the input units.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the start positions of sparse data.
 * @param pSparseEnd Pointer to the end positions of sparse data.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 */
__global__ void kLoadIndexedSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t warpId = tid / cData._warpSize;
    
    if (warpId < batch)
    {
        uint32_t pos = position + warpId;
        pos = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos] : pos];
        
        uint64_t start = pSparseStart[pos] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos];
        NNFloat w = (pDataWeight != NULL) ? pDataWeight[pos] : (NNFloat)1.0;
        uint64_t offset = warpId * stride;
        
        while (start < end)
        {
            uint64_t pos2 = offset + pSparseIndex[start];
            pUnit[pos2] = w;
            start += cData._warpSize;
        }
    }
}

/**
 * @brief Function to load indexed sparse input units.
 *
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride of the input.
 * @param pUnit Pointer to the input units.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the start positions of sparse data.
 * @param pSparseEnd Pointer to the end positions of sparse data.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 */
void kLoadIndexedSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseInputUnit failed");
    
    kLoadIndexedSparseInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadIndexedSparseInputUnit_kernel");
}

/**
 * \brief CUDA kernel for loading sparse analog input units.
 *
 * \tparam T Data type for sparse data.
 *
 * \param position The starting position.
 * \param batch The batch size.
 * \param stride The stride.
 * \param pUnit Pointer to the input unit.
 * \param pSparseStart Pointer to the sparse start array.
 * \param pSparseEnd Pointer to the sparse end array.
 * \param pSparseIndex Pointer to the sparse index array.
 * \param pDataWeight Pointer to the data weight array.
 * \param pSparseData Pointer to the sparse data array.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS()
kLoadSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset = pos * stride;

        for (uint64_t i = start; i < end; i += cData._warpSize)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * data;
        }
    }
}

/**
 * @brief Load sparse analog input unit data into GPU memory.
 *
 * @tparam T Type of the sparse data.
 * @param position Starting position of the data.
 * @param batch Number of elements to load.
 * @param stride Stride between elements.
 * @param pUnit Pointer to the GPU memory for storing the loaded data.
 * @param pSparseStart Pointer to the start indices of the sparse data.
 * @param pSparseEnd Pointer to the end indices of the sparse data.
 * @param pSparseIndex Pointer to the indices of the sparse data.
 * @param pDataWeight Pointer to the data weights of the sparse data.
 * @param pSparseData Pointer to the sparse data.
 */
template<typename T>
void kLoadSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemcpy(pUnit, &NNFloat(0), (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat), hipMemcpyHostToDevice);
    RTERROR(status, "kLoadSparseAnalogInputUnit failed");

    kLoadSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadSparseAnalogInputUnit_kernel");
}

/**
 * @brief CUDA kernel for loading indexed sparse analog input units.
 *
 * @tparam T The type of the sparse data.
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride value.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the sparse start array.
 * @param pSparseEnd Pointer to the sparse end array.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 * @param pSparseData Pointer to the sparse data array.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS()
kLoadIndexedSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* __restrict__ pUnit, uint32_t* __restrict__ pIndex, uint64_t* __restrict__ pSparseStart, uint64_t* __restrict__ pSparseEnd, uint32_t* __restrict__ pSparseIndex, NNFloat* __restrict__ pDataWeight, T* __restrict__ pSparseData)
{
    constexpr uint32_t warpSize = cData._warpSize;
    constexpr uint32_t warpMask = cData._warpMask;

    uint32_t pos = threadIdx.x / warpSize;
    if (pos < batch)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset = pos * stride;

        #pragma unroll
        for (uint64_t i = start; i < end; i += warpSize)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * data;
        }
    }
}

/**
 * @brief Load sparse analog input units using index-based lookup.
 *
 * This function loads sparse analog input units from the given sparse data and index arrays
 * into the output array. It sets the memory of the output array to zero before loading the values.
 *
 * @tparam T The data type of the sparse data.
 * @param position The starting position of the sparse input units to load.
 * @param batch The number of sparse input units to load.
 * @param stride The stride between consecutive elements in the output array.
 * @param pUnit Pointer to the output array for the sparse input units.
 * @param pIndex Pointer to the index array for the sparse input units.
 * @param pSparseStart Pointer to the array of starting indices for the sparse data.
 * @param pSparseEnd Pointer to the array of ending indices for the sparse data.
 * @param pSparseIndex Pointer to the array of indices for the sparse data.
 * @param pDataWeight Pointer to the array of data weights for the sparse data.
 * @param pSparseData Pointer to the sparse data array.
 */
template<typename T>
void kLoadIndexedSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<size_t>(batch) * static_cast<size_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "hipMemset failed");

    kLoadIndexedSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadIndexedSparseAnalogInputUnit_kernel");

    status = hipGetLastError();
    RTERROR(status, "Kernel execution failed");
}

/**
 * @brief CUDA kernel for loading sparse denoised input units.
 *
 * @param position    The starting position in the batch.
 * @param batch       The number of elements to process in the batch.
 * @param stride      The stride value.
 * @param pUnit       Pointer to the output units.
 * @param pSparseStart Pointer to the start indices of sparse data for each position.
 * @param pSparseEnd   Pointer to the end indices of sparse data for each position.
 * @param pSparseIndex Pointer to the indices of sparse data.
 * @param pDataWeight  Pointer to the weight data.
 * @param pRandom      Pointer to the random data.
 */
__global__ void kLoadSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride,
                                                    NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd,
                                                    uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t pos = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset = pos * stride;

        __shared__ NNFloat sDataWeight[BLOCK_SIZE];
        if (pDataWeight != NULL && threadIdx.x < cData._warpSize)
            sDataWeight[threadIdx.x] = pDataWeight[pos1];
        
        uint64_t loopEnd = end - cData._warpSize;

        #pragma unroll
        for (; start < loopEnd; start += cData._warpSize)
        {
            NNFloat value = pRandom[start];
            uint64_t pos2 = offset + pSparseIndex[start];

            NNFloat mask = __ballot_sync(FULL_MASK, value >= cData._denoising_p);
            if (threadIdx.x % cData._warpSize == 0)
                pUnit[pos2] = w * (__popc(mask) == cData._warpSize ? 1.0f : 0.0f);
        }

        if (start < end)
        {
            NNFloat value = pRandom[start];
            uint64_t pos2 = offset + pSparseIndex[start];
            NNFloat mask = __ballot_sync(FULL_MASK, value >= cData._denoising_p);
            if (threadIdx.x % cData._warpSize == 0 && (start < loopEnd || threadIdx.x < (end - start)))
                pUnit[pos2] = w * (__popc(mask) == cData._warpSize ? 1.0f : 0.0f);
        }
    }
}

/**
 * @brief Loads sparse denoised input units into a CUDA device memory.
 *
 * This function sets the specified memory region to zero and then launches a CUDA kernel
 * to load sparse denoised input units into the memory.
 *
 * @param position       Starting position of the input units.
 * @param batch          Number of input units to load.
 * @param stride         Stride of the input units.
 * @param pUnit          Pointer to the device memory for storing the input units.
 * @param pSparseStart   Pointer to the start indices of the sparse input units.
 * @param pSparseEnd     Pointer to the end indices of the sparse input units.
 * @param pSparseIndex   Pointer to the indices of the sparse input units.
 * @param pDataWeight    Pointer to the data weights of the sparse input units.
 * @param pRandom        Pointer to random values for sparse input generation.
 */
void kLoadSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    size_t unitSize = (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat);

    hipError_t status = hipMemsetAsync(pUnit, 0, unitSize);
    RTERROR(status, "hipMemsetAsync failed");

    kLoadSparseDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    CUDA_CHECK_ERROR("kLoadSparseDenoisedInputUnit_kernel launch failed");

    hipDeviceSynchronize();
}

/**
 * @brief Kernel for loading indexed sparse denoised input units.
 *
 * @param position       Starting position in the input data.
 * @param batch          Number of input samples to process.
 * @param stride         Stride between consecutive input samples.
 * @param pUnit          Pointer to the output unit array.
 * @param pIndex         Pointer to the index array.
 * @param pSparseStart   Pointer to the start indices of sparse data.
 * @param pSparseEnd     Pointer to the end indices of sparse data.
 * @param pSparseIndex   Pointer to the indices of sparse data.
 * @param pDataWeight    Pointer to the weight array.
 * @param pRandom        Pointer to the random values array.
 */
__global__ void LAUNCH_BOUNDS()
kLoadIndexedSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride,
                                           NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart,
                                           uint64_t* pSparseEnd, uint32_t* pSparseIndex,
                                           NNFloat* pDataWeight, NNFloat* pRandom)
{
    const uint32_t warpIndex = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (warpIndex < batch)
    {
        const uint32_t globalIndex = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[warpIndex + position] : warpIndex + position];
        const uint64_t warpStart = pSparseStart[globalIndex] + (threadIdx.x & cData._warpMask);
        const uint64_t warpEnd = pSparseEnd[globalIndex];
        const NNFloat weight = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[globalIndex] : (NNFloat)1.0);
        const uint64_t offset = warpIndex * stride;

        for (uint64_t i = warpStart; i < warpEnd; i += cData._warpSize)
        {
            const NNFloat value = pRandom[i];
            const uint64_t unitIndex = offset + pSparseIndex[i];

            if (value >= cData._denoising_p)
            {
                pUnit[unitIndex] = weight;
            }
        }
    }
}

/**
 * @brief Loads denoised input units from indexed sparse data into a CUDA memory buffer.
 *
 * @param position The starting position of the input units.
 * @param batch The number of input units to load.
 * @param stride The stride of the input units.
 * @param pUnit Pointer to the CUDA memory buffer to store the loaded units.
 * @param pIndex Pointer to the index array containing the indices of the input units.
 * @param pSparseStart Pointer to the array of starting positions for each index in the sparse data.
 * @param pSparseEnd Pointer to the array of ending positions for each index in the sparse data.
 * @param pSparseIndex Pointer to the array of indices for each data entry in the sparse data.
 * @param pDataWeight Pointer to the weight values associated with each data entry in the sparse data.
 * @param pRandom Pointer to the array of random values.
 */
void kLoadIndexedSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count * getGpu()._warpSize + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseDenoisedInputUnit failed");
    kLoadIndexedSparseDenoisedInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    LAUNCHERROR("kLoadIndexedSparseDenoisedInputUnit_kernel");
}

/**
 * @brief Kernel function for loading sparse analog denoised input units.
 *
 * @tparam T The type of sparse data.
 * @param position The position parameter.
 * @param batch The batch parameter.
 * @param stride The stride parameter.
 * @param pUnit Pointer to the unit data.
 * @param pSparseStart Pointer to the sparse start data.
 * @param pSparseEnd Pointer to the sparse end data.
 * @param pSparseIndex Pointer to the sparse index data.
 * @param pDataWeight Pointer to the data weight.
 * @param pSparseData Pointer to the sparse data.
 * @param pRandom Pointer to the random data.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS()
kLoadSparseAnalogDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom)
{
    uint32_t pos = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset = pos * stride;

        uint32_t* pIndex = pSparseIndex + start;
        T* pData = pSparseData + start;

        __shared__ NNFloat randomShared[cData._warpSize];
        if (threadIdx.x < cData._warpSize)
            randomShared[threadIdx.x] = pRandom[start];

        for (; start < end; start += cData._warpSize)
        {
            NNFloat value = randomShared[threadIdx.x & cData._warpMask];
            uint64_t pos2 = offset + pIndex[threadIdx.x];
            T data = pData[threadIdx.x];
            if (value >= cData._denoising_p)
                pUnit[pos2] = w * data;
        }
    }
}

/**
 * @brief Loads sparse analog denoised input unit.
 *
 * This function loads the sparse analog denoised input unit using CUDA.
 * It sets the memory to zero using CUDA memsetAsync for asynchronous memory initialization,
 * and then launches the kernel to load the sparse data into the input unit.
 *
 * @tparam T         The type of the sparse data.
 * @param position   The starting position in the input unit.
 * @param batch      The batch size.
 * @param stride     The stride of the input unit.
 * @param pUnit      Pointer to the input unit.
 * @param pSparseStart Pointer to the start indices of the sparse data.
 * @param pSparseEnd Pointer to the end indices of the sparse data.
 * @param pSparseIndex Pointer to the indices of the sparse data.
 * @param pDataWeight Pointer to the weights of the sparse data.
 * @param pSparseData Pointer to the sparse data.
 * @param pRandom    Pointer to the random data.
 */
template<typename T>
void kLoadSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T *pSparseData, NNFloat* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipError_t status = hipMemsetAsync(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat), stream);
    RTERROR(status, "kLoadSparseAnalogDenoisedInputUnit failed");

    kLoadSparseAnalogDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock, 0, stream>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    LAUNCHERROR("kLoadSparseAnalogDenoisedInputUnit_kernel");

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
}

/**
 * @brief Kernel function for loading indexed sparse analog denoised input units.
 *
 * @tparam T The data type for the sparse data.
 * @param position The starting position of the batch.
 * @param batch The number of elements in the batch.
 * @param stride The stride between elements.
 * @param pUnit Pointer to the output unit data.
 * @param pIndex Pointer to the index data.
 * @param pSparseStart Pointer to the start indices of sparse data.
 * @param pSparseEnd Pointer to the end indices of sparse data.
 * @param pSparseIndex Pointer to the indices of sparse data.
 * @param pDataWeight Pointer to the weight data.
 * @param pSparseData Pointer to the sparse data.
 * @param pRandom Pointer to the random data.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, const uint64_t* pSparseStart, const uint64_t* pSparseEnd, const uint32_t* pSparseIndex, const NNFloat* pDataWeight, const T* pSparseData, const NNFloat* pRandom)
{
    uint32_t pos = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset = pos * stride;

        __shared__ NNFloat sDataWeight[BLOCK_SIZE];
        if (threadIdx.x < BLOCK_SIZE)
            sDataWeight[threadIdx.x] = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        __syncthreads();

        for (uint64_t i = start; i < end; i += cData._warpSize)
        {
            NNFloat values[WARP_SIZE];
            T data[WARP_SIZE];
            uint64_t pos2[WARP_SIZE];

            #pragma unroll
            for (int j = 0; j < WARP_SIZE; ++j)
            {
                if (i + j < end)
                {
                    values[j] = pRandom[i + j];
                    pos2[j] = offset + pSparseIndex[i + j];
                    data[j] = pSparseData[i + j];
                }
            }

            #pragma unroll
            for (int j = 0; j < WARP_SIZE; ++j)
            {
                if (i + j < end)
                {
                    NNFloat value = values[j];
                    uint64_t pos2_1 = pos2[j];
                    T data1 = data[j];
                    NNFloat output1 = w * data1;

                    if (value < cData._denoising_p)
                        output1 = 0.0;

                    pUnit[pos2_1] = output1;
                }
            }
        }
    }
}

/**
 * @brief Loads indexed sparse analog denoised input units into a CUDA device memory array.
 *
 * This function initializes the device memory array `pUnit` with indexed sparse analog denoised input units
 * based on the provided parameters. It performs memory initialization asynchronously using `hipMemsetAsync`.
 * Then, it copies the necessary data from the host to the device memory using `hipMemcpyAsync`.
 * Finally, it launches the kernel `kLoadIndexedSparseAnalogDenoisedInputUnit_kernel` to compute the input units.
 *
 * @tparam T The type of the sparse data.
 * @param position The starting position of the input units.
 * @param batch The number of input units to process.
 * @param stride The stride of the input units.
 * @param pUnit Pointer to the device memory array for storing the input units.
 * @param pIndex Pointer to the device memory array containing the indices.
 * @param pSparseStart Pointer to the device memory array containing the sparse start indices.
 * @param pSparseEnd Pointer to the device memory array containing the sparse end indices.
 * @param pSparseIndex Pointer to the device memory array containing the sparse indices.
 * @param pDataWeight Pointer to the device memory array containing the data weights.
 * @param pSparseData Pointer to the device memory array containing the sparse data.
 * @param pRandom Pointer to the device memory array containing random values.
 */
template<typename T>
void kLoadIndexedSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipMemsetAsync(pUnit, 0, batch * stride * sizeof(NNFloat));
    hipError_t status = hipGetLastError();
    RTERROR(status, "hipMemsetAsync failed");

    hipStream_t memoryStream;
    hipStreamCreate(&memoryStream);

    hipMemcpyAsync(pUnit, pUnit, batch * stride * sizeof(NNFloat), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pIndex, pIndex, batch * sizeof(uint32_t), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pSparseStart, pSparseStart, batch * sizeof(uint64_t), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pSparseEnd, pSparseEnd, batch * sizeof(uint64_t), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pSparseIndex, pSparseIndex, batch * stride * sizeof(uint32_t), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pDataWeight, pDataWeight, batch * stride * sizeof(NNFloat), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pSparseData, pSparseData, batch * stride * sizeof(T), hipMemcpyDeviceToDevice, memoryStream);
    hipMemcpyAsync(pRandom, pRandom, batch * stride * sizeof(NNFloat), hipMemcpyDeviceToDevice, memoryStream);

    kLoadIndexedSparseAnalogDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock, 0, memoryStream>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    hipError_t kernelLaunchStatus = hipGetLastError();
    RTERROR(kernelLaunchStatus, "Kernel launch error: kLoadIndexedSparseAnalogDenoisedInputUnit_kernel");

    hipStreamDestroy(memoryStream);
    hipDeviceSynchronize();
}

/**
 * @brief Loads input units from input data into the specified output array.
 *
 * @tparam T The data type of the input data.
 * @param position The position within the input data.
 * @param stride The stride value.
 * @param pUnit The output array for storing the input units.
 * @param pInputData The input data array.
 */
template<typename T>
__global__ void kLoadInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pInputData)
{
    uint32_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : (blockIdx.x + position);
        uint32_t soffset = pos1 * stride + pos;
        uint32_t doffset = blockIdx.x * stride + pos;
        pUnit[doffset] = pInputData[soffset];
    }
}

/**
 * @brief CUDA kernel to load and normalize input units.
 *
 * This kernel loads input units from device memory and applies normalization
 * by scaling the data and subtracting a constant value.
 *
 * @param position The starting position of the input units.
 * @param stride The stride between input units.
 * @param pUnit Pointer to the output array of normalized units.
 * @param pData Pointer to the input data array.
 */
__global__ void kLoadNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData)
{
    uint32_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;

    if (pos < stride)
    {
        uint32_t pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position;
        uint32_t soffset = pos1 * stride + pos;
        uint32_t doffset = blockIdx.x * stride + pos;
        pUnit[doffset] = static_cast<NNFloat>(pData[soffset]) * static_cast<NNFloat>(1.0 / 256.0) - static_cast<NNFloat>(0.5);
    }
}

/**
 * @brief CUDA kernel to load normalized input units.
 *
 * @param position The position of the input unit.
 * @param stride The stride between input units.
 * @param pUnit Pointer to the output unit array.
 * @param pData Pointer to the input data array.
 */
__global__ void kLoadNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {

        uint32_t pos1 = cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position;
        uint64_t soffset = pos1 * stride + pos;
        uint64_t doffset = blockIdx.x * stride + pos;

        char dataValue = pData[soffset];
        pUnit[doffset] = static_cast<NNFloat>(dataValue) * (1.0f / 128.0f);
    }
}

/**
 * @brief Loads input unit data from the given pData array into the pUnit array.
 *
 * @tparam T The data type of the pData array.
 * @param position The position of the input unit.
 * @param batch The batch size.
 * @param stride The stride of the data.
 * @param pUnit Pointer to the output unit array.
 * @param pData Pointer to the input data array.
 */
template <typename T>
void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    hipDeviceSynchronize(); // Wait for the kernel to finish execution
    checkCudaErrors(hipGetLastError()); // Check for any errors during kernel execution
}

/**
 * @brief Specialization of kLoadInputUnit for unsigned char data type.
 *
 * @param position The position of the input unit.
 * @param batch The batch size.
 * @param stride The stride of the data.
 * @param pUnit Pointer to the output unit array.
 * @param pData Pointer to the input data array.
 */
template <>
void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, unsigned char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}

/**
 * @brief Specialization of kLoadInputUnit for char data type.
 *
 * @param position The position of the input unit.
 * @param batch The batch size.
 * @param stride The stride of the data.
 * @param pUnit Pointer to the output unit array.
 * @param pData Pointer to the input data array.
 */
template <>
void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}

/**
 * @brief CUDA kernel to load indexed input units into the output unit array.
 *
 * @param position The starting position of the input units.
 * @param stride The stride value.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the input data array.
 */
template<typename T>
__global__ void kLoadIndexedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData)
{
    size_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        size_t blockIdxPos = blockIdx.x + position;
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[blockIdxPos] : blockIdxPos];
        size_t soffset = pos1 * stride + pos;
        size_t doffset = blockIdx.x * stride + pos;
        pUnit[doffset] = pData[soffset];
    }
}

/**
 * @brief CUDA kernel to load indexed and normalized input units into the neural network.
 *
 * @param position Starting position in the input data.
 * @param stride Stride size for accessing the input units.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the input data array.
 */
__global__ void kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position];
        uint64_t soffset = pos1 * stride + pos;
        uint64_t doffset = blockIdx.x * stride + pos;
        NNFloat dataValue = static_cast<NNFloat>(pData[soffset]) * static_cast<NNFloat>(1.0 / 256.0) - static_cast<NNFloat>(0.5);
        pUnit[doffset] = dataValue;
    }
}

/**
 * @brief CUDA kernel to load indexed and normalized input units into the neural network.
 *
 * @param position Starting position in the input data.
 * @param stride Stride size for accessing the input units.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the input data array.
 */
__global__ void kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData)
{
    uint64_t pos = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position];
        uint64_t soffset = pos1 * stride + pos;
        uint64_t doffset = blockIdx.x * stride + pos;
        NNFloat dataValue = static_cast<NNFloat>(static_cast<unsigned char>(pData[soffset])) * static_cast<NNFloat>(1.0 / 128.0);
        pUnit[doffset] = dataValue;
    }
}

/**
 * @brief Loads indexed input units into the neural network.
 *
 * @tparam T Type of the input data.
 * @param position Starting position in the input data.
 * @param batch Number of batches.
 * @param stride Stride size for accessing the input units.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the input data array.
 */
template<typename T>
void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData)
{
    const uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    const dim3 grid(batch, (stride + threadsPerBlock - 1) / threadsPerBlock);
    kLoadIndexedInputUnit_kernel<<<grid, threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);
    LAUNCHERROR("kLoadIndexedInputUnit_kernel");
}

/**
 * @brief Loads indexed input units from unsigned char data into NNFloat array.
 *
 * @param position The position of the input unit.
 * @param batch The number of input units to load.
 * @param stride The stride between consecutive input units.
 * @param pUnit Pointer to the NNFloat array to store the loaded input units.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the unsigned char data.
 */
template<> void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData)
{
    uint32_t numBlocks = (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    dim3 grid(batch, numBlocks);

    kLoadIndexedNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    LAUNCHERROR("kLoadIndexedNormalizedInputUnit_kernel");
}

/**
 * @brief Loads indexed input units from char data into NNFloat array.
 *
 * @param position The position of the input unit.
 * @param batch The number of input units to load.
 * @param stride The stride between consecutive input units.
 * @param pUnit Pointer to the NNFloat array to store the loaded input units.
 * @param pIndex Pointer to the index array.
 * @param pData Pointer to the char data.
 */
template<> void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData)
{
    uint32_t numBlocks = (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    dim3 grid(batch, numBlocks);

    kLoadIndexedNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    LAUNCHERROR("kLoadIndexedNormalizedInputUnit_kernel");
}

/**
 * @brief CUDA kernel to add bias values to a unit array.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride value.
 * @param size Total number of elements in the unit array.
 */
__global__ void kAddBias_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t size)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] += pBias[bpos];
    }
}

/**
 * @brief Function to launch the kAddBias_kernel CUDA kernel.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride value.
 * @param batch Batch size.
 */
void kAddBias(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint32_t size = stride * batch;
    uint32_t threadsPerBlock = 256;  // Choose an appropriate value based on the GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kAddBias_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error in kAddBias: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}

/**
 * @brief CUDA kernel to add dual biases to the unit array.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param size     Total size of the unit array.
 */
__global__ void kAddDualBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] += pBias1[bpos] + pBias2[bpos];
    }
}

/**
 * @brief Adds dual biases to the unit array using CUDA.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param batch    Number of batches in the unit array.
 */
void kAddDualBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kAddDualBias_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kAddDualBias_kernel");
}

/**
 * @brief CUDA kernel to add triple biases to the unit array.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param pBias3   Pointer to the third bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param size     Total size of the unit array.
 */
__global__ void kAddTripleBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] += pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

/**
 * @brief Adds triple biases to the unit array using CUDA.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param pBias3   Pointer to the third bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param batch    Number of batches in the unit array.
 */
void kAddTripleBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kAddTripleBias_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kAddTripleBias_kernel");
}

/**
 * @brief CUDA kernel to add quad biases to the unit array.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param pBias3   Pointer to the third bias array.
 * @param pBias4   Pointer to the fourth bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param size     Total size of the unit array.
 */
__global__ void kAddQuadBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] += pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

/**
 * @brief Adds quad biases to the unit array using CUDA.
 *
 * @param pUnit    Pointer to the unit array.
 * @param pBias1   Pointer to the first bias array.
 * @param pBias2   Pointer to the second bias array.
 * @param pBias3   Pointer to the third bias array.
 * @param pBias4   Pointer to the fourth bias array.
 * @param stride   Stride value for indexing the bias arrays.
 * @param batch    Number of batches in the unit array.
 */
void kAddQuadBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kAddQuadBias_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kAddQuadBias_kernel");
}

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 600)
constexpr uint32_t MAXSPARSE = SM_6X_MAXSPARSE;
constexpr uint32_t MAXSPARSEANALOG = SM_6X_MAXSPARSEANALOG;
#elif defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 500)
constexpr uint32_t MAXSPARSE = SM_5X_MAXSPARSE;
constexpr uint32_t MAXSPARSEANALOG = SM_5X_MAXSPARSEANALOG;
#else
constexpr uint32_t MAXSPARSE = SM_3X_MAXSPARSE;
constexpr uint32_t MAXSPARSEANALOG = SM_3X_MAXSPARSEANALOG;
#endif
