#include "hip/hip_runtime.h"
#include "GpuTypes.h"
#include "Types.h"
#include <limits>

/**
 * @brief Constant data stored on the GPU.
 */
static __constant__ GpuData cData;

/**
 * @brief Copies the GPU data to the constant symbol cData.
 */
void SetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

/**
 * @brief Copies the GPU data from the constant symbol cData.
 */
void GetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy from cData failed");
}

/**
 * @brief Calculates the number of blocks required for the given size.
 *
 * @param size The total number of elements.
 * @return The number of blocks needed to process the elements.
 */
uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

/**
 * @brief CUDA kernel to scale and bias an array in parallel.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
__global__ void kScaleAndBias_kernel(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint64_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        NNFloat value = pData[offset];
        pData[offset] = scale * value - bias;
    }
}

/**
 * @brief Applies scale and bias to an array using CUDA.
 *
 * @param pData Pointer to the data array.
 * @param size Size of the data array.
 * @param scale Scale factor to apply to each element.
 * @param bias Bias value to subtract from each element.
 */
void kScaleAndBias(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kScaleAndBias_kernel<<<blocks, threadsPerBlock>>>(pData, size, scale, bias);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a unit by assigning bias values in parallel.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearUnit_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;
    if (pos < size)
    {
        pUnit[pos] = pBias[bpos];
    }
}

/**
 * @brief Clears a unit using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias Pointer to the bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearUnit(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = 256; // Choose an appropriate value based on your GPU architecture
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias, stride, size);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        // Handle or report the CUDA kernel launch error
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to clear a dual source unit by summing bias values in parallel.
 *
 * This kernel function sets the values of a dual source unit by summing the corresponding bias values from two different bias arrays.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param size Size of the unit array.
 */
__global__ void kClearDualSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos = pos % stride;

    if (pos < size)
    {
        pUnit[pos] = pBias1[bpos] + pBias2[bpos];
    }
}

/**
 * @brief Clears a dual source unit by summing bias values in parallel.
 *
 * This function clears a dual source unit by summing the bias values from two different bias arrays in parallel using CUDA.
 *
 * @param pUnit Pointer to the unit array.
 * @param pBias1 Pointer to the first bias array.
 * @param pBias2 Pointer to the second bias array.
 * @param stride Stride of the unit array.
 * @param batch Batch size.
 */
void kClearDualSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);

    uint32_t threadsPerBlock = 256;

    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    kClearDualSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief CUDA kernel to compute the triple source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearTripleSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

/**
 * @brief Compute the triple source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearTripleSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearTripleSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSource_kernel");
}

/**
 * @brief CUDA kernel to compute the quad source unit values.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param size     Size of the output unit array
 */
__global__ void kClearQuadSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        uint32_t bpos = pos % stride;
        pUnit[pos] = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

/**
 * @brief Compute the quad source unit values using CUDA.
 *
 * @param pUnit    Pointer to the output unit array
 * @param pBias1   Pointer to the first bias array
 * @param pBias2   Pointer to the second bias array
 * @param pBias3   Pointer to the third bias array
 * @param pBias4   Pointer to the fourth bias array
 * @param stride   Stride of the bias arrays
 * @param batch    Batch size
 */
void kClearQuadSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size = static_cast<uint64_t>(stride) * static_cast<uint64_t>(batch);
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    kClearQuadSourceUnit_kernel<<<blocks, threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSource_kernel");
}

/**
 * @brief CUDA kernel to load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
__global__ void kLoadSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1];
        uint64_t end = pSparseEnd[pos1];

        __shared__ NNFloat weight;
        if (threadIdx.x == 0)
        {
            weight = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        }
        __syncthreads();

        uint64_t offset = pos * stride;

        for (uint64_t i = threadIdx.x; i < (end - start); i += blockDim.x)
        {
            uint64_t pos2 = offset + pSparseIndex[start + i];
            pUnit[pos2] = weight;
        }
    }
}

/**
 * @brief Load sparse input units into a dense unit matrix.
 *
 * @param position      The starting position of the batch.
 * @param batch         The number of batches to process.
 * @param stride        The stride of the unit matrix.
 * @param pUnit         Pointer to the unit matrix.
 * @param pSparseStart  Pointer to the array containing the start positions of sparse data.
 * @param pSparseEnd    Pointer to the array containing the end positions of sparse data.
 * @param pSparseIndex  Pointer to the array containing the sparse indices.
 * @param pDataWeight   Pointer to the array containing the data weights (optional).
 */
void kLoadSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t threadsPerBlock = getGpu()._threadsPerBlock;
    uint32_t blocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<uint64_t>(batch) * static_cast<uint64_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseInputUnit failed");
    kLoadSparseInputUnit_kernel<<<blocks, threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadSparseInputUnit_kernel");
}

/**
 * @brief CUDA kernel for loading indexed sparse input units.
 *
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride of the input.
 * @param pUnit Pointer to the input units.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the start positions of sparse data.
 * @param pSparseEnd Pointer to the end positions of sparse data.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 */
__global__ void kLoadIndexedSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t warpId = tid / cData._warpSize;
    
    if (warpId < batch)
    {
        uint32_t pos = position + warpId;
        pos = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos] : pos];
        
        uint64_t start = pSparseStart[pos] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos];
        NNFloat w = (pDataWeight != NULL) ? pDataWeight[pos] : (NNFloat)1.0;
        uint64_t offset = warpId * stride;
        
        while (start < end)
        {
            uint64_t pos2 = offset + pSparseIndex[start];
            pUnit[pos2] = w;
            start += cData._warpSize;
        }
    }
}

/**
 * @brief Function to load indexed sparse input units.
 *
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride of the input.
 * @param pUnit Pointer to the input units.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the start positions of sparse data.
 * @param pSparseEnd Pointer to the end positions of sparse data.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 */
void kLoadIndexedSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseInputUnit failed");
    
    kLoadIndexedSparseInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadIndexedSparseInputUnit_kernel");
}

/**
 * \brief CUDA kernel for loading sparse analog input units.
 *
 * \tparam T Data type for sparse data.
 *
 * \param position The starting position.
 * \param batch The batch size.
 * \param stride The stride.
 * \param pUnit Pointer to the input unit.
 * \param pSparseStart Pointer to the sparse start array.
 * \param pSparseEnd Pointer to the sparse end array.
 * \param pSparseIndex Pointer to the sparse index array.
 * \param pDataWeight Pointer to the data weight array.
 * \param pSparseData Pointer to the sparse data array.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS()
kLoadSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < batch)
    {
        uint32_t pos1 = pos + position;
        pos1 = cData._bShuffleIndices ? cData._pShuffleIndex[pos1] : pos1;
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset = pos * stride;

        for (uint64_t i = start; i < end; i += cData._warpSize)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * data;
        }
    }
}

/**
 * @brief Load sparse analog input unit data into GPU memory.
 *
 * @tparam T Type of the sparse data.
 * @param position Starting position of the data.
 * @param batch Number of elements to load.
 * @param stride Stride between elements.
 * @param pUnit Pointer to the GPU memory for storing the loaded data.
 * @param pSparseStart Pointer to the start indices of the sparse data.
 * @param pSparseEnd Pointer to the end indices of the sparse data.
 * @param pSparseIndex Pointer to the indices of the sparse data.
 * @param pDataWeight Pointer to the data weights of the sparse data.
 * @param pSparseData Pointer to the sparse data.
 */
template<typename T>
void kLoadSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemcpy(pUnit, &NNFloat(0), (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat), hipMemcpyHostToDevice);
    RTERROR(status, "kLoadSparseAnalogInputUnit failed");

    kLoadSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadSparseAnalogInputUnit_kernel");
}

/**
 * @brief CUDA kernel for loading indexed sparse analog input units.
 *
 * @tparam T The type of the sparse data.
 * @param position The starting position.
 * @param batch The number of batches.
 * @param stride The stride value.
 * @param pUnit Pointer to the output unit array.
 * @param pIndex Pointer to the index array.
 * @param pSparseStart Pointer to the sparse start array.
 * @param pSparseEnd Pointer to the sparse end array.
 * @param pSparseIndex Pointer to the sparse index array.
 * @param pDataWeight Pointer to the data weight array.
 * @param pSparseData Pointer to the sparse data array.
 */
template<typename T>
__global__ void LAUNCH_BOUNDS()
kLoadIndexedSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* __restrict__ pUnit, uint32_t* __restrict__ pIndex, uint64_t* __restrict__ pSparseStart, uint64_t* __restrict__ pSparseEnd, uint32_t* __restrict__ pSparseIndex, NNFloat* __restrict__ pDataWeight, T* __restrict__ pSparseData)
{
    constexpr uint32_t warpSize = cData._warpSize;
    constexpr uint32_t warpMask = cData._warpMask;

    uint32_t pos = threadIdx.x / warpSize;
    if (pos < batch)
    {
        uint32_t pos1 = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start = pSparseStart[pos1] + (threadIdx.x & warpMask);
        uint64_t end = pSparseEnd[pos1];
        NNFloat w = (pDataWeight != nullptr) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset = pos * stride;

        #pragma unroll
        for (uint64_t i = start; i < end; i += warpSize)
        {
            uint64_t pos2 = offset + pSparseIndex[i];
            T data = pSparseData[i];
            pUnit[pos2] = w * data;
        }
    }
}

/**
 * @brief Load sparse analog input units using index-based lookup.
 *
 * This function loads sparse analog input units from the given sparse data and index arrays
 * into the output array. It sets the memory of the output array to zero before loading the values.
 *
 * @tparam T The data type of the sparse data.
 * @param position The starting position of the sparse input units to load.
 * @param batch The number of sparse input units to load.
 * @param stride The stride between consecutive elements in the output array.
 * @param pUnit Pointer to the output array for the sparse input units.
 * @param pIndex Pointer to the index array for the sparse input units.
 * @param pSparseStart Pointer to the array of starting indices for the sparse data.
 * @param pSparseEnd Pointer to the array of ending indices for the sparse data.
 * @param pSparseIndex Pointer to the array of indices for the sparse data.
 * @param pDataWeight Pointer to the array of data weights for the sparse data.
 * @param pSparseData Pointer to the sparse data array.
 */
template<typename T>
void kLoadIndexedSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last = position + batch;
    uint32_t count = last - position;
    uint32_t blocks = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status = hipMemset(pUnit, 0, static_cast<size_t>(batch) * static_cast<size_t>(stride) * sizeof(NNFloat));
    RTERROR(status, "hipMemset failed");

    kLoadIndexedSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadIndexedSparseAnalogInputUnit_kernel");

    status = hipGetLastError();
    RTERROR(status, "Kernel execution failed");
}
