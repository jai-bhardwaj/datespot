#include "hip/hip_runtime.h"
#include "../engine/bitonic.h"
#include "Output.h"
#include <limits>

/**
 * @brief CUDA kernel for calculating the output.
 *
 * @param pOutputBuffer Pointer to the output buffer.
 * @param pKeyBuffer Pointer to the key buffer.
 * @param pValueBuffer Pointer to the value buffer.
 * @param batch The number of batches.
 * @param width The width of the data.
 * @param widthPadding The width padding.
 * @param k The value of k.
 */
__global__ void kCalculateOutput_kernel(float* pOutputBuffer, float* pKeyBuffer, unsigned int* pValueBuffer,
                                        unsigned int batch, unsigned int width, unsigned int widthPadding,
                                        unsigned int k)
{
    constexpr int sharedMemorySize = 160 * 4;
    __shared__ volatile float sKey[sharedMemorySize];
    __shared__ volatile unsigned int sValue[sharedMemorySize];

    unsigned int dataWidth = width - widthPadding;
    unsigned int pos = (blockIdx.x * blockDim.x + threadIdx.x) >> 5;
    unsigned int tgx = threadIdx.x & 31;

    if (pos < batch)
    {
        float* pOutput = pOutputBuffer + pos * width;
        unsigned int offset = threadIdx.x >> 5;
        volatile float* psKey = &sKey[sharedMemorySize * offset];
        volatile unsigned int* psValue = &sValue[sharedMemorySize * offset];

        constexpr float minValue = -std::numeric_limits<float>::max();
        constexpr int bufferSizeLimit = 128;
        constexpr int warpSize = 32;
        constexpr int fullWarpSize = 32;
        constexpr int halfWarpSize = 16;

        float k0 = minValue, k1 = minValue, k2 = minValue, k3 = minValue;
        unsigned int v0 = 0, v1 = 0, v2 = 0, v3 = 0;

        unsigned int wpos = tgx;
        if (wpos < dataWidth)
        {
            k0 = pOutput[wpos];
            v0 = wpos;
        }
        wpos += warpSize;
        if (wpos < dataWidth)
        {
            k1 = pOutput[wpos];
            v1 = wpos;
        }
        wpos += warpSize;
        if (wpos < dataWidth)
        {
            k2 = pOutput[wpos];
            v2 = wpos;
        }
        wpos += warpSize;
        if (wpos < dataWidth)
        {
            k3 = pOutput[wpos];
            v3 = wpos;
        }

        unsigned int rpos = fullWarpSize;
        int bufferSize = 0;

        while (rpos < dataWidth)
        {
            unsigned int wpos = rpos + tgx;
            float key = minValue;
            unsigned int value = wpos;
            if (wpos < dataWidth)
            {
                key = pOutput[wpos];
            }

            unsigned int count = __ballot_sync(0xffffffff, key > minValue);

            if (key > minValue)
            {
                unsigned int mask = 0xffffffff >> (warpSize - tgx);
                unsigned int offset = __popc_sync(0xffffffff, count & mask) + bufferSize;
                psKey[offset] = key;
                psValue[offset] = value;
            }

            bufferSize += __popc_sync(0xffffffff, count);

            if (bufferSize >= bufferSizeLimit)
            {
                k2 = psKey[tgx + 2 * warpSize];
                v2 = psValue[tgx + 2 * warpSize];
                k3 = psKey[tgx + 3 * warpSize];
                v3 = psValue[tgx + 3 * warpSize];

                BITONICSORT256_256(psKey, psValue, tgx);

                minValue = __shfl_down_sync(0xffffffff, k3, halfWarpSize);

                bufferSize -= bufferSizeLimit;
                if (tgx < bufferSize)
                {
                    psKey[tgx] = psKey[tgx + bufferSizeLimit];
                    psValue[tgx] = psValue[tgx + bufferSizeLimit];
                }
            }

            rpos += warpSize;
        }

        if (bufferSize > 0 || dataWidth <= bufferSizeLimit)
        {
            k2 = minValue;
            k3 = minValue;
            v2 = 0;
            v3 = 0;

            if (tgx < bufferSize)
            {
                k2 = psKey[tgx];
                v2 = psValue[tgx];
            }
            if (tgx + warpSize < bufferSize)
            {
                k3 = psKey[tgx + warpSize];
                v3 = psValue[tgx + warpSize];
            }

            BITONICSORT256_256(psKey, psValue, tgx);
        }

        float* pKey = pKeyBuffer + pos * k;
        unsigned int* pValue = pValueBuffer + pos * k;
        wpos = tgx;
        if (wpos < k)
        {
            pKey[wpos] = k0;
            pValue[wpos] = v0;
        }
        wpos += warpSize;
        if (wpos < k)
        {
            pKey[wpos] = k1;
            pValue[wpos] = v1;
        }
        wpos += warpSize;
        if (wpos < k)
        {
            pKey[wpos] = k2;
            pValue[wpos] = v2;
        }
        wpos += warpSize;
        if (wpos < k)
        {
            pKey[wpos] = k3;
            pValue[wpos] = v3;
        }
    }
}

/**
 * @brief Calculates the output using CUDA.
 *
 * @param pOutput Pointer to the output.
 * @param pKey Pointer to the key.
 * @param pValue Pointer to the value.
 * @param batch The number of batches.
 * @param width The width of the data.
 * @param widthPadding The width padding.
 * @param k The value of k.
 */
void kCalculateOutput(float* pOutput, float* pKey, unsigned int* pValue, unsigned int batch, unsigned int width,
                      unsigned int widthPadding, unsigned int k)
{
    unsigned int blocks = (batch + 3) / 4;
    kCalculateOutput_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, widthPadding, k);
    LAUNCHERROR("kCalculateOutput_kernel");
}

